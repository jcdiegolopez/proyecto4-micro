#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define DATA_LENGTH 55       // Número de filas
#define THRESHOLD_BPM 60     // Umbral para BPM bajo
#define WINDOW_SIZE 3        // Tamaño de la ventana para promedio móvil

// Kernel: Normalización
__global__ void normalize(float *data, float *normalized, float min, float max, int length) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < length) {
        normalized[idx] = (data[idx] - min) / (max - min);
    }
}

// Kernel: Promedio móvil
__global__ void movingAverage(float *data, float *smoothed, int length, int window_size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < length) {
        float sum = 0.0;
        int count = 0;
        for (int j = -window_size / 2; j <= window_size / 2; j++) {
            int neighbor = idx + j;
            if (neighbor >= 0 && neighbor < length) {
                sum += data[neighbor];
                count++;
            }
        }
        smoothed[idx] = sum / count;
    }
}

// Kernel: Detección de cambios bruscos
__global__ void detectAbruptChanges(float *data, bool *abrupt_changes, int length, float threshold) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < length - 1) {
        abrupt_changes[idx] = fabsf(data[idx + 1] - data[idx]) > threshold;
    }
}

// Kernel: Identificación de BPM bajo
__global__ void detectLowBPM(float *bpm, bool *low_bpm, int length, float threshold) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < length) {
        low_bpm[idx] = bpm[idx] < threshold;
    }
}

// Kernel: Clasificación del estado final
__global__ void classifySleep(bool *low_bpm, bool *abrupt_changes, bool *sleep_result, int length) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < length) {
        sleep_result[idx] = low_bpm[idx] || abrupt_changes[idx];
    }
}

int main() {
    // Variables para almacenar los datos
    float bpm[DATA_LENGTH], accel_y[DATA_LENGTH];

    // Abrir el archivo binario
    FILE *file = fopen("data.bin", "rb");
    if (!file) {
        printf("Error al abrir el archivo data.bin\n");
        return 1;
    }

    // Leer los datos del archivo binario
    size_t read_count = fread(bpm, sizeof(float), DATA_LENGTH, file);
    read_count += fread(accel_y, sizeof(float), DATA_LENGTH, file);
    fclose(file);

    // Verificar si los datos fueron leídos correctamente
    if (read_count != DATA_LENGTH * 2) {
        printf("Error al leer los datos del archivo. Elementos leídos: %lu\n", read_count);
        return 1;
    }

    // Imprimir los datos leídos
    printf("Datos leídos del archivo:\n");
    for (int i = 0; i < DATA_LENGTH; i++) {
        printf("Fila %d: BPM=%.2f, Aceleración Y=%.2f\n", i, bpm[i], accel_y[i]);
    }

    // Reservar memoria en GPU
    float *d_bpm, *d_accel_y, *d_normalized, *d_smoothed;
    bool *d_low_bpm, *d_abrupt_changes, *d_sleep_result;
    hipMalloc((void **)&d_bpm, DATA_LENGTH * sizeof(float));
    hipMalloc((void **)&d_accel_y, DATA_LENGTH * sizeof(float));
    hipMalloc((void **)&d_normalized, DATA_LENGTH * sizeof(float));
    hipMalloc((void **)&d_smoothed, DATA_LENGTH * sizeof(float));
    hipMalloc((void **)&d_low_bpm, DATA_LENGTH * sizeof(bool));
    hipMalloc((void **)&d_abrupt_changes, DATA_LENGTH * sizeof(bool));
    hipMalloc((void **)&d_sleep_result, DATA_LENGTH * sizeof(bool));

    // Copiar datos a GPU
    hipMemcpy(d_bpm, bpm, DATA_LENGTH * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_accel_y, accel_y, DATA_LENGTH * sizeof(float), hipMemcpyHostToDevice);

    // Normalizar datos
    float accel_min = -5.0;  // Valor mínimo estimado de aceleración
    float accel_max = 5.0;   // Valor máximo estimado de aceleración
    int threads_per_block = 256;
    int blocks_per_grid = (DATA_LENGTH + threads_per_block - 1) / threads_per_block;
    normalize<<<blocks_per_grid, threads_per_block>>>(d_accel_y, d_normalized, accel_min, accel_max, DATA_LENGTH);

    // Suavizar con promedio móvil
    movingAverage<<<blocks_per_grid, threads_per_block>>>(d_normalized, d_smoothed, DATA_LENGTH, WINDOW_SIZE);

    // Detectar cambios bruscos
    detectAbruptChanges<<<blocks_per_grid, threads_per_block>>>(d_smoothed, d_abrupt_changes, DATA_LENGTH, 0.1);

    // Detectar BPM bajo
    detectLowBPM<<<blocks_per_grid, threads_per_block>>>(d_bpm, d_low_bpm, DATA_LENGTH, THRESHOLD_BPM);

    // Clasificar estado de sueño
    classifySleep<<<blocks_per_grid, threads_per_block>>>(d_low_bpm, d_abrupt_changes, d_sleep_result, DATA_LENGTH);

    // Copiar resultados al host
    bool *low_bpm = (bool *)malloc(DATA_LENGTH * sizeof(bool));
    bool *abrupt_changes = (bool *)malloc(DATA_LENGTH * sizeof(bool));
    bool *sleep_result = (bool *)malloc(DATA_LENGTH * sizeof(bool));
    hipMemcpy(low_bpm, d_low_bpm, DATA_LENGTH * sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(abrupt_changes, d_abrupt_changes, DATA_LENGTH * sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(sleep_result, d_sleep_result, DATA_LENGTH * sizeof(bool), hipMemcpyDeviceToHost);

    // Imprimir resultados
    printf("\nResultados de clasificación de sueño:\n");
    for (int i = 0; i < DATA_LENGTH; i++) {
        printf("Fila %d: BPM=%.2f, Aceleración Y=%.2f, Normalizada=%.2f, BPM Bajo=%s, Cambio Brusco=%s, Resultado Final=%s\n",
       i, bpm[i], accel_y[i], accel_y[i] / accel_max, low_bpm[i] ? "Sí" : "No", abrupt_changes[i] ? "Sí" : "No", sleep_result[i] ? "Dormido" : "Despierto");

    }

    // Liberar memoria
    free(low_bpm);
    free(abrupt_changes);
    free(sleep_result);
    hipFree(d_bpm);
    hipFree(d_accel_y);
    hipFree(d_normalized);
    hipFree(d_smoothed);
    hipFree(d_low_bpm);
    hipFree(d_abrupt_changes);
    hipFree(d_sleep_result);

    return 0;
}
